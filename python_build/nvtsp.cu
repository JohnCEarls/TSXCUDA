#include "hip/hip_runtime.h"
/*=================================================================
 *
 *  nvtspmex.cu
 *  Author: Andrew Magis
 *  Calculate TSP scores on the GPU
 *  Inputs: Class 1 data, Class 2 data, N (size of cross-validation)
 *  Outputs: TSP primary scores, TSP secondary scores, TSP upper and lower bounds for CV
 *
 *
 *=================================================================*/

#include <math.h>
//#include "mex.h"
#include <vector>
#include <iostream>
#include "nvtsp.cuh"
using std::cout;
using std::endl;
#ifndef __NVTSP_CU_
#define __NVTSP_CU_
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#endif
void DisplayDeviceProperties(int device) {

    hipDeviceProp_t deviceProp;
    memset(&deviceProp, 0, sizeof (deviceProp));
	
	printf("-----\n");
	
    if (hipSuccess == hipGetDeviceProperties(&deviceProp, device)) {
		printf("Device Name\t\t\t\t%s\n", deviceProp.name);
		printf("Total Global Memory\t\t\t%ld KB\n",deviceProp.totalGlobalMem / 1024);
		printf("Maximum threads per block\t\t%d\n", deviceProp.maxThreadsPerBlock);
		
    } else {
        printf("\n%s", hipGetErrorString(hipGetLastError()));
    }
	
	printf("------\n");				
		
}

#define THREADS 16
#define ABSMACRO(X) ( ((X)<0) ? (-(X)) : (X) )
#define ABSBINARYMACRO(X) ( ((X)<0) ? (0) : (1) )
#define MINMACRO(X,Y) ( ((X)<(Y)) ? (X) : (Y) )
#define MAXMACRO(X,Y) ( ((X)>(Y)) ? (X) : (Y) )

__global__ void tspKernel(float *d_class1, float *d_class2, unsigned int n1, unsigned int n2, unsigned int m, unsigned int cvn, float *primary, float *secondary, float *lower, float *upper, int *vote) {
	
    float class1_score = 0.f;
	float class2_score = 0.f;
	float class1_rank = 0.f;
	float class2_rank = 0.f;	
	float temp_lower1, temp_lower2;
	float temp_upper1, temp_upper2;
	
	float n1_invert = __fdividef(1.f, (float)n1);
	float n2_invert = __fdividef(1.f, (float)n2);
	
	//We are only building a diagonal matrix here, so return if I am part of the diagonal
	//or below the diagonal
	if ((blockIdx.x*blockDim.x+threadIdx.x) > (blockIdx.y*blockDim.y+threadIdx.y)) {
	
		//Pointers to correct memory location for class1
		float *data1 = &d_class1[(blockIdx.x*blockDim.x + threadIdx.x)];
		float *data2 = &d_class1[(blockIdx.y*blockDim.y + threadIdx.y)];

		for (int i = 0; i < n1*m; i+=m) {
			//if (data1[i] <= data2[i]) {
			//	class1_score += 1.f;
			//}	
			class1_score += signbit(data1[i]-data2[i]);
			class1_rank += (float)(data1[i]-data2[i]);
		}
		temp_lower1 = __fdividef(class1_score-(float)cvn, (float)(n1-cvn));
		temp_upper1 = __fdividef(class1_score, (float)(n1-cvn));
		class1_score = class1_score * n1_invert;
		class1_rank = class1_rank * n1_invert;

		//Pointers to correct memory location for class2
		data1 =  &d_class2[(blockIdx.x*blockDim.x + threadIdx.x)];
		data2 =  &d_class2[(blockIdx.y*blockDim.y + threadIdx.y)];
	
		for (int i = 0; i < n2*m; i+=m) {
			//if (data1[i] <= data2[i]) {
			//	class2_score += 1.f;
			//}		
			class2_score += signbit(data1[i]-data2[i]);
			class2_rank += (float)(data1[i]-data2[i]);
		}
		temp_lower2 = __fdividef(class2_score, (float)(n2-cvn));
		temp_upper2 = __fdividef(class2_score-(float)cvn, (float)(n2-cvn));
		class2_score = class2_score * n2_invert;
		class2_rank = class2_rank * n2_invert;
		
		temp_lower1 = ABSMACRO(temp_lower1 - class2_score);
		temp_lower2 = ABSMACRO(class1_score - temp_lower2); 
		temp_upper1 = ABSMACRO(temp_upper1 - class2_score);
		temp_upper2 = ABSMACRO(class1_score - temp_upper2);
	
	}

	//Write the result to global memory
	primary[(blockIdx.x*blockDim.x + threadIdx.x)*m + (blockIdx.y*blockDim.y + threadIdx.y)] = ABSMACRO(class1_score-class2_score);
	secondary[(blockIdx.x*blockDim.x + threadIdx.x)*m + (blockIdx.y*blockDim.y + threadIdx.y)] = ABSMACRO(class1_rank-class2_rank);
	lower[(blockIdx.x*blockDim.x + threadIdx.x)*m + (blockIdx.y*blockDim.y + threadIdx.y)] = MINMACRO(temp_lower1, temp_lower2);
	upper[(blockIdx.x*blockDim.x + threadIdx.x)*m + (blockIdx.y*blockDim.y + threadIdx.y)] = MAXMACRO(temp_upper1, temp_upper2);
	vote[(blockIdx.x*blockDim.x + threadIdx.x)*m + (blockIdx.y*blockDim.y + threadIdx.y)] = ABSBINARYMACRO(class1_score-class2_score);

}
/**

If I am lucky this is the only bit I'm going to have to rewrite
**/
void nvwrapper( std::vector<double> & data1, int dsSize1, std::vector<int> & classSizes1 ){
    std::vector<double> data = data1;
    int dsSize = dsSize1;
    std::vector<int> classSizes = classSizes1;
	
	//DisplayDeviceProperties(0);

/**	//Time the execution of this function
	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);
    hipEventSynchronize(start_event);
    float time_run;**/
    // gonna have to pass this in as a parameter, but not interested in doing that  the moment.
	unsigned int cvn = 5;//(unsigned int)(cvn_temp[0]);
	printf("Size of cross-validation is is %u\n", cvn);

	//m is the number of rows (genes)
	//n is the number of chips (samples)
	unsigned int m1 = dsSize;
	unsigned int n1 = classSizes[0];
	unsigned int m2 = dsSize;
	unsigned int n2 = classSizes[1];
	
	//Create a padded m which is multiple of THREADS
	unsigned int m;
	if (m1 % THREADS == 0) {
		m = m1;
	} else {
		m = ((int)(m1 / THREADS) + 1) * THREADS;
	}
	printf("Class1 Ranks: [%d, %d] Class2 Ranks: [%d, %d]\n", m1, n1, m2, n2);
	printf("Thread Dimension: %d Padded length: %d\n", THREADS, m);

    /**Five outputs required (TSP primary scores, TSP secondary scores, lower bounds, upper bounds, vote)
    **/
	// Create an mxArray for the output data - this is automatically zeroed out
    //near as I can tell we are creating 5 ngenes x ngenes arrays, four floats and one int
    /**
    May be able to get away with using vectors
    **/

    float TSPPrimaryScores[m1][m1];// = new float*[m1];
    float TSPSecondaryScores[m1][m1];// = new float*[m1];
    float lowerbounds[m1][m1];// float*[m1];
    float upperbounds[m1][m1];// = new float*[m1];
    float vote[m1][m1];// = new float*[m1];
	
    unsigned long int class1_size = m*n1 * sizeof(float);
	unsigned long int class2_size = m*n2 * sizeof(float);
	unsigned long int result_size = m*m * sizeof(float);
	
	//Allocate space on the GPU to store the input data
	float *d_class1, *d_class2;
    hipMalloc( (void**)&d_class1, class1_size );
    hipMalloc( (void**)&d_class2, class2_size ); 
             printf("Memory allocating failure on the GPU.");
			
	//Allocate space on the GPU to store the output data
	float *d_s1, *d_s2, *d_s3, *d_s4, *d_s5;
    //debugging
    float d_test = 1.0;
    d_s1 = d_s2 = d_s3 = d_s4 = d_s5 = &d_test;	
    
    if( ( hipMalloc( (void**)&d_s1, result_size )  != hipSuccess )
    || ( hipMalloc( (void**)&d_s2, result_size )  != hipSuccess )
    || ( hipMalloc( (void**)&d_s3, result_size )  != hipSuccess )
    || ( hipMalloc( (void**)&d_s4, result_size )  != hipSuccess )
    || ( hipMalloc( (void**)&d_s5, result_size )  != hipSuccess )){
			cout << "Memory allocating failure on the GPU." << endl;
            if(d_s1 == &d_test){
                cout << "failed on";
                cout << __LINE__ << endl;
            }
            if(d_s2 == &d_test){
                cout << "failed on";
                cout << __LINE__ << endl;
            }if(d_s3 == &d_test){
                cout << "failed on";
                cout << __LINE__ << endl;
            }if(d_s4 == &d_test){
                cout << "failed on";
                cout << __LINE__ << endl;
            }if(d_s5 == &d_test){
                cout << "failed on";
                cout << __LINE__ << endl;
            }
    }
			
	//Reallocate space for the data on the host with zeroed out padding
	float *h_class1, *h_class2, *h_s1, *h_s2, *h_s3, *h_s4, *h_s5;
	if ((hipHostMalloc((void**)&h_class1, class1_size) != hipSuccess) 
	|| (hipHostMalloc((void**)&h_class2, class2_size) != hipSuccess)
	|| (hipHostMalloc((void**)&h_s1, result_size) != hipSuccess) 
	|| (hipHostMalloc((void**)&h_s2, result_size) != hipSuccess) 
	|| (hipHostMalloc((void**)&h_s3, result_size) != hipSuccess) 
	|| (hipHostMalloc((void**)&h_s4, result_size) != hipSuccess) 
	|| (hipHostMalloc((void**)&h_s5, result_size) != hipSuccess) ){
	    cout <<	"Memory allocating failure on the host." << endl;	
    }
		
	//Zero out the memory on the host
	memset(h_class1, 0, class1_size);
	memset(h_class2, 0, class2_size);
	//Copy over data to new padded array location on host
    //k back to near as I can tell
    //this appears to be copying the data into the GPU
    //prob a good time to make our dataFloatArray
    //may not have to do this if I make the double vector a float vector.
    float mtemp_trough[data.size()];
    for (int i = 0; i<data.size();i++){
               mtemp_trough[i] = (float)data.at(i);
   }
    float *mtemp = mtemp_trough;
	float *temp = h_class1;
	for (int i = 0; i < n1; i++) {
		memcpy(temp, mtemp, m1*sizeof(float));
		mtemp += m1;
		temp += m;
	}	
	temp = h_class2;
    cout << __LINE__ << endl;
	for (int i = 0; i < n2; i++) {
		memcpy(temp, mtemp, m1*sizeof(float));
		mtemp += m1;
		temp += m;
	}		
							
	//Copy data to the GPU
	if ( (hipMemcpy(d_class1, h_class1, class1_size, hipMemcpyHostToDevice) != hipSuccess) || (hipMemcpy(d_class2, h_class2, class2_size, hipMemcpyHostToDevice) != hipSuccess)){
		cout << "Error copying memory to the GPU.";
    }
		
	//Set the dimension of the blocks and grids
	dim3 dimBlock(THREADS, THREADS);
	dim3 dimGrid(m/THREADS, m/THREADS);
    cout << __LINE__ << endl;
	
	printf("Scheduling [%d %d] threads in [%d %d] blocks\n", THREADS, THREADS, m/THREADS, m/THREADS);
	//tspKernel<<<dimGrid, dimBlock>>>(d_class1, d_class2, n1, n2, m, cvn, d_s1, d_s2, d_s3, d_s4, (int*)d_s5);
	hipDeviceSynchronize();
    cout << __LINE__ << endl;
		
	//Copy the memory back
	if ((hipMemcpy(h_s1, d_s1, result_size, hipMemcpyDeviceToHost) != hipSuccess) ||
	 (hipMemcpy(h_s2, d_s2, result_size, hipMemcpyDeviceToHost) != hipSuccess) 
	|| (hipMemcpy(h_s3, d_s3, result_size, hipMemcpyDeviceToHost) != hipSuccess) 
	|| (hipMemcpy(h_s4, d_s4, result_size, hipMemcpyDeviceToHost) != hipSuccess) 
	|| (hipMemcpy(h_s5, d_s5, result_size, hipMemcpyDeviceToHost) != hipSuccess) )
		cout << "Error copying memory from the GPU.";	
		
	float *gpu_output1 = h_s1, *gpu_output2 = h_s2, *gpu_output3 = h_s3, *gpu_output4 = h_s4, *gpu_output5 = h_s5;
    if (sizeof(float) == sizeof(int)){
        cout << sizeof(float);
        cout << "float is equal to int" << endl;
    } else {
        cout << "float neq int"<<endl;
    }
    /**
	//Finally, copy the padded array data into the output matrix
	for (int i = 0; i < m1; i++) {
		memcpy(TSPPrimaryScores, gpu_output1, m1*sizeof(float));
		memcpy(TSPSecondaryScores, gpu_output2, m1*sizeof(float));
		memcpy(lower_bounds, gpu_output3, m1*sizeof(float));
		memcpy(upper_bounds, gpu_output4, m1*sizeof(float));
		memcpy(vote, gpu_output5, m1*sizeof(float));			
		TSPPrimaryScores += m1; TSPSecondaryScores += m1; lower_bounds += m1; upper_bounds += m1; vote += m1;
		gpu_output1 += m; gpu_output2 += m; gpu_output3 += m; gpu_output4 += m; gpu_output5 += m;
	}**/		
	
    /**
    TODO
    Memory cleanup and pushing data into output
    

    **/
	/**
	hipEventRecord(stop_event, 0);
	hipEventSynchronize(stop_event); // block until the event is actually recorded
	hipEventElapsedTime(&time_run, start_event, stop_event);
	printf("Finished running nvTSP in %.6f seconds\n", time_run / 1000.0);
	hipEventRecord(start_event, 0);
    hipEventSynchronize(start_event);	
	**/
	//Clear up memory on the device
	hipFree(d_class1);
	hipFree(d_class2);
	hipFree(d_s1); 
	hipFree(d_s2);
	hipFree(d_s3);
	hipFree(d_s4);
	hipFree(d_s5);
    /**
	//Clear up memory on the host
	hipHostFree(h_class1);
	hipHostFree(h_class2);
	hipHostFree(h_s1); 
	hipHostFree(h_s2);
	hipHostFree(h_s3);
	hipHostFree(h_s4);	
	hipHostFree(h_s5);**/
    hipDeviceSynchronize();
   

    cout << __LINE__ << endl;
/**    for(int i=0; i<m1 ; i++){
        delete [] TSPPrimaryScores[i];
        delete [] TSPSecondaryScores[i];
        delete [] lower_bounds[i];
        delete [] upper_bounds[i];
        delete [] vote[i];
   }
        delete [] TSPPrimaryScores;
        delete [] TSPSecondaryScores;
        delete [] lower_bounds;
        delete [] upper_bounds;
        delete [] vote;
        d_class1 = d_class2 = d_s1 = d_s2 = d_s3 = d_s4 = d_s5 =
h_class1 = 
h_class2 = 
h_s1 = 
h_s2 = 
h_s3 = 
h_s4 = 
h_s5= NULL;
        TSPPrimaryScores = 
        TSPSecondaryScores= 
        lower_bounds= 
        upper_bounds= 
        vote = NULL;
**/
    hipDeviceReset();
  
}

